#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void baseline_add(float* A, float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

extern "C"
void run_baseline(int N) {
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threads(256);
    dim3 blocks((N + 255) / 256);

    baseline_add<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
