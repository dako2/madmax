// shared_mem.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void shared_mem_add(float* A, float* B, float* C, int N) {
    extern __shared__ float tile[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (idx < N) {
        // Split shared memory for A and B
        float* tileA = tile;
        float* tileB = tile + blockDim.x;

        // Load to shared memory
        tileA[tid] = A[idx];
        tileB[tid] = B[idx];

        __syncthreads();

        // Compute and write result
        C[idx] = tileA[tid] + tileB[tid];
    }
}

extern "C"
void run_shared_mem(int N) {
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threads(256);
    dim3 blocks((N + 255) / 256);
    size_t sharedMemSize = 2 * threads.x * sizeof(float);

    shared_mem_add<<<blocks, threads, sharedMemSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
