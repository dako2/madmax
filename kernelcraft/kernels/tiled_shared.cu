#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Tiled kernel using shared memory
__global__ void tiled_shared_add(float* A, float* B, float* C, int N) {
    extern __shared__ float tile[];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float* tileA = tile;
    float* tileB = tile + blockDim.x;

    for (int i = idx; i < N; i += stride) {
        // Load tile
        tileA[tid] = A[i];
        tileB[tid] = B[i];

        __syncthreads();

        // Compute
        C[i] = tileA[tid] + tileB[tid];

        __syncthreads();  // optional if no shared reuse
    }
}

extern "C"
void run_tiled_shared(int N) {
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = min((N + threads - 1) / threads, 1024);
    size_t sharedMemSize = 2 * threads * sizeof(float);

    tiled_shared_add<<<blocks, threads, sharedMemSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
