// tiled.cu
#include <hip/hip_runtime.h>
#include <stdio.h>

// Tiled vector add: multiple elements per thread
__global__ void tiled_add(float* A, float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }
}

extern "C"
void run_tiled(int N) {
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    if (blocks > 1024) blocks = 1024;  // Max grid size limit

    tiled_add<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}
